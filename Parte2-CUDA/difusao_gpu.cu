#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 256 // Tamanho da grade
#define T 500 // Número de iterações no tempo
#define D 0.1 // Coeficiente de difusão
#define DELTA_T 0.01
#define DELTA_X 1.0


__global__ void diff_eq(double *C, double *C_new, double *output, double *output_difmedio, int i){
		 	int x_position = threadIdx.x + blockIdx.x * blockDim.x;
			double up, down, left, right;
			int j;


			// calculo dos valores do halo do stencil
			up = (blockIdx.x > 0) ? C[threadIdx.x + (blockIdx.x - 1) * N] : 0.;
			down = (blockIdx.x < N-1) ? C[threadIdx.x + (blockIdx.x + 1) * N] : 0.;
			left = (threadIdx.x > 0) ? C[(threadIdx.x - 1) + blockIdx.x * N] : 0.;
			right = (threadIdx.x < N-1) ? C[(threadIdx.x + 1) + blockIdx.x * N] : 0.;

			//atualizacao da matriz com os novos valores
			C_new[x_position] = C[x_position] + D * DELTA_T * (
			(up + down + left + right - 4 * C[x_position]) / (DELTA_X * DELTA_X)
			);

			__syncthreads();

			//calculo do difmedio aqui

			if (threadIdx.x == 0) {
					 output[blockIdx.x] = 0;
					 for (j = 0; j < N; j ++){
								output[blockIdx.x] += fabs(C_new[x_position + j] - C[x_position + j]);
					 }
			}
			__syncthreads();
			if (x_position == 0) {
				output_difmedio[i] = 0;
				for (j = 0; j < N; j ++){
					output_difmedio[i] += output[j];
					if (i%100 == 0) printf("%g", output[j]);

				}
      	if (i%100 == 0) printf("%g", output_difmedio[i]);
			}

			C[x_position] = C_new[x_position];


}



int main(void)
{
	double *h_C = (double *)malloc(N*N*sizeof(double));
	double *h_C_new = (double *)malloc(N*N*sizeof(double));
	double *h_output = (double *)malloc(N*N*sizeof(double));
	double *h_output_difmedio = (double*)malloc(T*sizeof(double));

	double *d_C, *d_C_new, *d_output, *d_output_difmedio;

	// inicializacao da matriz
	for(int i=0; i< N*N; i++){
			 h_C[i] = 0.;
			 h_C_new[i] =0.;
	}

	int meio = ((N/2) * N) + N/2;

	h_C[meio] = 1.0;

	//alocacoes dos valores na GPU
	size_t size = N*N*sizeof(double);
	size_t prov = N *sizeof(double);

	hipMalloc(&d_C, size);
	hipMalloc(&d_C_new, size);
	hipMalloc(&d_output, prov);
	hipMalloc(&d_output_difmedio, T*sizeof(double));

	hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
	hipMemcpy(d_C_new, h_C_new, size, hipMemcpyHostToDevice);



	// T iteracoes do stencil, sendo cada iteracao realizada por um kernel
	for(int k=0;k<T; k++) {
			 diff_eq<<<N,N>>>(d_C, d_C_new, d_output, d_output_difmedio, k);
	}
	hipDeviceSynchronize();


	// Copia dos valores da GPU de volta para a CPU

	hipMemcpy(h_output, d_C, size, hipMemcpyDeviceToHost);
	hipMemcpy(h_output_difmedio, d_output_difmedio, T*sizeof(double), hipMemcpyDeviceToHost);

	printf("\n\nConcentração do meio: %g\n", h_output[meio]);
  for(int i=0;i<T;i++){
				if (i%100 == 0) {
					printf("interacao %d - diferenca = %g\n", i, h_output_difmedio[i]/((N)*(N)));
				}
	}
	return 0;
}